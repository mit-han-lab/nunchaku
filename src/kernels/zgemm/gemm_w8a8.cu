#include "zgemm.h"
#include "gemm_w8a8.cuh"

namespace nunchaku::kernels {

void quantize_w8a8_act(Tensor input, Tensor output, Tensor oscales, bool fuse_glu) {
    using GEMM = GEMM_W8A8;

    int M = input.numel() / input.shape[-1];
    int K = input.shape[-1];

    assert(output.dtype() == Tensor::INT8);
    assert(output.numel() / output.shape[-1] == M);
    assert(output.shape[-1] == fuse_glu ? K / 2 : K);

    assert(isTypeMatch<GEMM::half_t>(oscales.dtype()));
    assert(oscales.numel() == M * 1);

    auto launch = [&]<bool FUSE_GLU>() {
        using kernel = GEMM::quantize_w8a8_act_kernel<FUSE_GLU>;

        assert(kernel::check(M, K));
        dim3 grid  = kernel::gridSize(M, K);
        dim3 block = kernel::blockSize(M, K);

        auto func =
            invoke_kernel<kernel, const GEMM::half_t *, GEMM::packed_act_t *, GEMM::packed_ascale_t *, int, bool>;

        checkCUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(func), hipFuncAttributeMaxDynamicSharedMemorySize, 92160));

        func<<<grid, block, kernel::smemSize(M, K)>>>(input.data_ptr<GEMM::half_t>(),
                                                      output.data_ptr<GEMM::packed_act_t>(),
                                                      oscales.data_ptr<GEMM::packed_ascale_t>(),
                                                      K,
                                                      false);
        checkCUDA(hipGetLastError());
    };

    if (fuse_glu) {
        launch.template operator()<true>();
    } else {
        launch.template operator()<false>();
    }
}

void gemm_w8a8(Tensor act,     // [M, K]
               Tensor wgt,     // [N, K]
               Tensor out,     // [M, N]
               Tensor ascales, // [1, M]
               Tensor wscales, // [1, N]
               Tensor bias) {
    using GEMM = GEMM_W8A8;

    int M = act.numel() / act.shape[-1];
    int N = wgt.shape[0];
    int K = act.shape[-1];
    assert(K == wgt.shape[1]);

    int actualM = 0;
    int actualN = 0;
    if (out.valid()) {
        actualM = out.numel() / out.shape[-1];
        actualN = out.shape[-1];

        assert(actualM <= M && M - actualM < GEMM::BLOCK_M);
        assert(actualN <= N && N - actualN < GEMM::BLOCK_N);
    }

    auto launch = [&]<typename Epilogue>(Epilogue::Arguments args) {
        dim3 grid(M / GEMM::BLOCK_M, N / GEMM::BLOCK_N);

        bool swapBlockMN = M > N * 2;
        if (swapBlockMN) {
            std::swap(grid.x, grid.y);
        }

        invoke_kernel<GEMM::gemm_w8a8_kernel<Epilogue>>
            <<<grid, GEMM::WARP_SIZE * GEMM::NUM_WARPS>>>(act.data_ptr<GEMM::packed_act_t>(),
                                                          wgt.data_ptr<GEMM::packed_wgt_t>(),
                                                          ascales.data_ptr<GEMM::packed_ascale_t>(),
                                                          wscales.data_ptr<GEMM::packed_wscale_t>(),
                                                          // out.valid() ? out.data_ptr<GEMM::half_t>() : nullptr,
                                                          M,
                                                          N,
                                                          K,
                                                          args,
                                                          swapBlockMN,
                                                          false);
        checkCUDA(hipGetLastError());
    };

    auto launch_bias = [&]<typename NextEpilogue>(NextEpilogue::Arguments nextArgs) {
        if (!bias.valid()) {
            return launch.template operator()<NextEpilogue>(nextArgs);
        }

        assert(bias.numel() == N);

        // append EpilgoueNop to workaround mismatched memory layout of std::tuple between device and host code on
        // Windows
        // ** sizeof(std::tuple<std::tuple<int>>) == 8 on device **
        using Epilogue = GEMM::EpilogueCombination<GEMM::EpilogueBias<true, false>, NextEpilogue, GEMM::EpilogueNop>;
        return launch.template operator()<Epilogue>({GEMM::EpilogueBias<true, false>::Arguments{
                                                         .bias = bias.data_ptr<GEMM::packed_wscale_t>(),
                                                     },
                                                     nextArgs,
                                                     {}});
    };

    launch_bias.template operator()<GEMM::EpilogueDefault>(GEMM::EpilogueDefault::Arguments{
        .out     = out.data_ptr<GEMM::half_t>(),
        .actualM = actualM,
        .actualN = actualN,
    });
}

#if 0
void gemm_w8a8_fuse_litela(
    Tensor act,      // [B, (M), K]
    Tensor wgt,      // [N, K]
    Tensor out_q,    // [B, (M), N / 3]
    Tensor out_vk,   // [B, num_heads, head_dim + 1, head_dim]
    Tensor ascales,  // [1, M]
    Tensor wscales   // [1, N]
) {
    using GEMM = GEMM_W8A8;
    using Epilogue = GEMM::EpilogueLiteLA;

    int M = act.numel() / act.shape[-1];
    int N = wgt.shape[0];
    int K = act.shape[-1];
    assert(K == wgt.shape[1]);

    assert(out_vk.ndims() == 4);
    assert(out_vk.shape[2] == Epilogue::LITELA_HEAD_DIM + 1);
    assert(out_vk.shape[3] == Epilogue::LITELA_HEAD_DIM);
    assert(out_vk.shape[1] * Epilogue::LITELA_HEAD_DIM * 3 == N);

    int batch_size = out_vk.shape[0];
    int num_heads = out_vk.shape[1];

    assert(M % batch_size == 0);
    int batch_m = M / batch_size;

    Epilogue::Arguments epilogueArgs;
    epilogueArgs.batch_m = act.shape[1];
    epilogueArgs.out_q = out_q.data_ptr<GEMM::half_t>();
    epilogueArgs.out_vk = out_vk.data_ptr<float>();

    checkCUDA(hipMemsetAsync(out_vk.data_ptr(), 0, out_vk.buffer->getSize()));

    auto func = invoke_kernel<GEMM::gemm_w8a8_kernel<Epilogue>,
        const GEMM::packed_act_t *,
        const GEMM::packed_wgt_t *,
        const GEMM::packed_ascale_t *,
        const GEMM::packed_wscale_t *,
        // GEMM::half_t *,
        int, int, int,
        Epilogue::Arguments,
        bool,
        bool>;

    checkCUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(func), hipFuncAttributeMaxDynamicSharedMemorySize, Epilogue::SHMEM_SIZE));

    dim3 grid(M / GEMM::BLOCK_M, N / GEMM::BLOCK_N);

    bool swapBlockMN = M > N * 2;
    if (swapBlockMN) {
        std::swap(grid.x, grid.y);
    }

    func<<<grid, GEMM::WARP_SIZE * GEMM::NUM_WARPS, Epilogue::SHMEM_SIZE>>>(
        act.data_ptr<GEMM::packed_act_t>(),
        wgt.data_ptr<GEMM::packed_wgt_t>(),
        ascales.data_ptr<GEMM::packed_ascale_t>(),
        wscales.data_ptr<GEMM::packed_wscale_t>(),
        // nullptr,
        M, N, K, epilogueArgs,
        swapBlockMN,
        false
    );
    checkCUDA(hipGetLastError());

    invoke_kernel<Epilogue::vk_mul_q_kernel><<<dim3(batch_m / 128, num_heads, batch_size), 128>>>(
        out_q.data_ptr<GEMM::half_t>(),
        out_vk.data_ptr<float>(),
        1e-6f
    );
    checkCUDA(hipGetLastError());
}
#endif

}; // namespace nunchaku::kernels
